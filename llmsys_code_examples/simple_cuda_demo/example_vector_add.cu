#include <hip/hip_runtime.h>
#include <iostream>
#include <sstream>
#include <fstream>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <vector>

using namespace std;
using std::generate;
using std::vector;

__global__ void VecAddKernel(int* A, int* B, int* C, int n) {
  // blockDim is size of block along x-axis
  // blockIdx is the index of the current thread's block
  // threadIdx is the index of the current thread within the block
  // Compute the global thread ID
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    // Calculate the addition of the ith element of A and B
    C[i] = A[i] + B[i];
  }
}


extern "C" {

void VecAddCPU(int* A, int* B, int* C, int n) {
  for(int i = 0; i < n; ++i) {
    C[i] = A[i] + B[i];
  }
}


void VecAddCUDA(int* Agpu, int* Bgpu, int* Cgpu, int n) {
  // In this example, we load the data into the GPU by Python codes.
  int threads_per_block = 256;
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  VecAddKernel<<<num_blocks, threads_per_block>>>(Agpu, Bgpu, Cgpu, n);
}


void VecAddCUDA2(int* Acpu, int* Bcpu, int* Ccpu, int n) {
  // In this example, we load the data into the GPU by C++ codes.
  int *dA, *dB, *dC;
  // Allocate device memory
  hipMalloc(&dA, n * sizeof(int));
  hipMalloc(&dB, n * sizeof(int));
  hipMalloc(&dC, n * sizeof(int));
  // Copy data from host memory to device memory
  hipMemcpy(dA, Acpu, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dB, Bcpu, n * sizeof(int), hipMemcpyHostToDevice);
  // Launch the CUDA kernel
  int threads_per_block = 256;
  int num_blocks = (n + threads_per_block - 1) / threads_per_block;
  VecAddKernel<<<num_blocks, threads_per_block>>>(dA, dB, dC, n);
  // Copy the result from device memory to host memory
  hipMemcpy(Ccpu, dC, n * sizeof(int), hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(dA); 
  hipFree(dB); 
  hipFree(dC);
}

}

// Check result on the CPU
void verify_result_vecadd(vector<int> &a, vector<int> &b, vector<int> &c, int N) {
  // For every element...
  for (int i = 0; i < N; i++) {
    // For every element in the row-column pair
    // Check against the CPU result
    if (a[i] + b[i] != c[i]) {
      printf("Error in (%d): %d + %d != %d\n", i, a[i], b[i], c[i]);
    }
    assert(a[i] + b[i] == c[i]);
  }
}

int main() {
  // length of the vector
  for (int i=0; i<1000000; i++){
    int n = 1024;

    // Host vectors
    vector<int> h_a(n);
    vector<int> h_b(n);
    vector<int> h_c(n);

    // Initialize matrices
    generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
    generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

    
    VecAddCUDA2(h_a.data(), h_b.data(), h_c.data(), n);
    

    hipDeviceSynchronize();

    // Check CUDA execution
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "Error: %s\n", hipGetErrorString(err));
      // Handle the error (e.g., by exiting the program)
    }

    // Check result
    verify_result_vecadd(h_a, h_b, h_c, n);

    cout << i << ":Vector add verified! COMPLETED SUCCESSFULLY\n";
  }
}